#include "hip/hip_runtime.h"
#include "curandom.hpp"

__constant__ seed_t RNG_G = (seed_t)(6364136223846793005ull);
__constant__ seed_t RNG_C = (seed_t)(1442695040888963407ull);
__constant__ seed_t RNG_P = (seed_t)(1) << 63;

__global__ void rnd_real_kernel(int n, seed_t *seeds, float *reals) {
  float inv_RNG_P = (float)(1.0) / (float)((seed_t)(1) << 63);
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    seeds[i] = (RNG_G * seeds[i] + RNG_C) % RNG_P;
    reals[i] = seeds[i] * inv_RNG_P;
  }
}
