#include "gpu_detect.hpp"
#include <hip/hip_runtime.h>

int cuda_get_num_gpus(){
    int count = 0;
    hipGetDeviceCount(&count);
    return count;
}