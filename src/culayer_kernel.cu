#include "hip/hip_runtime.h"
#include "culayer.hpp"
#include <stdio.h>

// https://developer.download.nvidia.com/assets/cuda/files/reduction.pdf

#define DIV_UP(x,y) (1 + ((x - 1) / y))


__constant__ seed_t RNG_G = (seed_t)(6364136223846793005ull);
__constant__ seed_t RNG_C = (seed_t)(1442695040888963407ull);
__constant__ seed_t RNG_P = (seed_t)(1) << 63;

__device__ __forceinline__ float cu_rnd_real(seed_t* seed) {
  float inv_RNG_P = (float)(1) / (float)(RNG_P);
  *seed = (RNG_G * *seed + RNG_C) % RNG_P;
  return (float)(*seed) * inv_RNG_P;
}

__global__ void particle_step_kernel(int n,
  Particle* particles,
  int steps,
  float const* const sigs_in,
  float const* const absorption_rates_in,
  float * const weights_absorbed_out,
  int min_index,
  int max_index,
  float dx)
{
  extern __shared__ float sdata[];

  int n_cells = max_index-min_index;

  float * const sigs = sdata;
  float * const absorption_rates = sdata + n_cells;
  float * const weights_absorbed = sdata + 2*n_cells;

  for (int j = 0; j < DIV_UP(n_cells, blockDim.x); j++){
    int cpy_ind = j*blockDim.x + threadIdx.x;
    if (cpy_ind < n_cells){
      sigs[cpy_ind] = sigs_in[cpy_ind];
      absorption_rates[cpy_ind] = absorption_rates_in[cpy_ind];
      weights_absorbed[cpy_ind] = 0;
    }
  }
  __syncthreads();

  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < n){
    Particle particle = particles[i];

    for (int step = 0; step < steps; step++){
      if (particle.index >= min_index && particle.index < max_index)
      {
        int local_index = particle.index - min_index;
        const float interaction_rate = 1.0 - absorption_rates[local_index];
        const float sig_a = sigs[local_index] * absorption_rates[local_index];
        const float sig_i = sigs[local_index] * interaction_rate;

        // calculate theoretic movement
        const float h = cu_rnd_real(&particle.seed);
        float di = MAXREAL;
        if (sig_i > EPS_PRECISION){
          // This should always be true
          di = -log(h) / sig_i;
        }

        // -- possible new cell --
        float mu_sign = copysignf(1.0, particle.mu);
        int index_new = __float2int_rn(mu_sign) + particle.index;
        float x_new_edge = particle.index * dx;
        if (mu_sign == 1){
          x_new_edge += dx;
        }

        float di_edge = MAXREAL;
        if (particle.mu < -EPS_PRECISION || EPS_PRECISION < particle.mu){
          di_edge = (x_new_edge - particle.x) / particle.mu;
        }

        if (di < di_edge) {
          /* move inside cell an draw new mu */
          index_new = particle.index;
          particle.x += di * particle.mu;
          particle.mu = 2 * cu_rnd_real(&particle.seed) - 1;
        } else {
          /* set position to border */
          di = di_edge;
          particle.x = x_new_edge;
        }

        // -- Calculate amount of absorbed energy --
        const float dw = (1 - expf(-sig_a * di)) * particle.wmc;

        /* Weight removed from particle is added to the layer */
        particle.wmc -= dw;
        atomicAdd(weights_absorbed + local_index, dw);
        particle.index = index_new;

        
      }
    }
    particles[i] = particle;
  }

  __syncthreads();
  for (int j = 0; j < DIV_UP(n_cells, blockDim.x); j++){
    int cpy_ind = j*blockDim.x + threadIdx.x;
    if (cpy_ind < n_cells){
      atomicAdd(weights_absorbed_out + cpy_ind, weights_absorbed[cpy_ind]);
    }
  }
}