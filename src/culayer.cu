#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "particle.hpp"
#include "culayer_kernel.hpp"
#include "culayer.hpp"
#include "gpu_errcheck.hpp"
#include <sys/time.h>
#include <math.h>

// https://devblogs.nvidia.com/using-shared-memory-cuda-cc/

#define DIV_UP(x,y) (1 + ((x - 1) / y))
#define FLOAT_CMP_PREC (1e-4)

void particle_sort(
  int n_in, 
  Particle* particles_in, 
  int* n_active, 
  Particle* particles_active,
  int* n_inactive,
  Particle* particles_inactive,
  int min_index,
  int max_index){

  for(int i = 0; i < n_in; i++)
  {
    if(particles_in[i].index < min_index || particles_in[i].index >= max_index)
    {
      particles_inactive[*n_inactive] = particles_in[i];
      (*n_inactive)++;
    }
    else
    {
      particles_active[*n_active] = particles_in[i];
      (*n_active)++;
    }
  }
}

void simulate(int n,
  Particle* particles,
  float const* const sigs,
  float const* const absorption_rates,
  float * const weights_absorbed,
  int min_index,
  int max_index,
  float dx)
  {
    int n_cells = max_index - min_index;
    
    int n_active = n;
    int n_inactive = 0;

    int steps = 325;

    Particle* particles_inactive = (Particle*) malloc(sizeof(Particle) * n);
    Particle* buffer = (Particle*) malloc(sizeof(Particle) * n);

    Particle* d_particles;
    gpu_errcheck( hipMalloc((void**)&d_particles, sizeof(Particle) * n) );

    float *d_sigs, *d_absorption_rates, *d_weights_absorbed;
    gpu_errcheck( hipMalloc((void**)&d_sigs, sizeof(float) * n_cells) );
    gpu_errcheck( hipMalloc((void**)&d_absorption_rates, sizeof(float) * n_cells) );
    gpu_errcheck( hipMalloc((void**)&d_weights_absorbed, sizeof(float) * n_cells) );

    gpu_errcheck( hipMemcpy(d_sigs, sigs, sizeof(float) * n_cells, hipMemcpyHostToDevice) );
    gpu_errcheck( hipMemcpy(d_absorption_rates, absorption_rates, sizeof(float) * n_cells, hipMemcpyHostToDevice) );
    gpu_errcheck( hipMemcpy(d_weights_absorbed, weights_absorbed, sizeof(float) * n_cells, hipMemcpyHostToDevice) );

    while(n_active > 0){
      gpu_errcheck( hipMemcpy(d_particles, particles, sizeof(Particle) * n_active, hipMemcpyHostToDevice) );

      printf("%d\n", n_active);
      particle_step_kernel<<<DIV_UP(n_active, 256), 256, sizeof(float) * 3 * n_cells >>>(
        n_active, d_particles, steps, d_sigs, d_absorption_rates, d_weights_absorbed, min_index, max_index, dx );
      gpu_errcheck( hipPeekAtLastError() );

      gpu_errcheck( hipMemcpy(particles, d_particles, sizeof(Particle) * n_active, hipMemcpyDeviceToHost) );
      {
        int n_active_new = 0;
        particle_sort(n_active, particles, &n_active_new, buffer, &n_inactive, particles_inactive, min_index, max_index);
        n_active = n_active_new;
        Particle* tmp_ptr = particles;
        particles = buffer;
        buffer = tmp_ptr;
      }
    }

    // printf("\n");
    gpu_errcheck( hipMemcpy(weights_absorbed, d_weights_absorbed, sizeof(float) * n_cells, hipMemcpyDeviceToHost) );    
  }